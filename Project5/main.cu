#include <iostream>
#include <hip/hip_runtime.h>
#include "matrix.h"
#include <chrono>

#define CUDA_CHECK(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            std::cerr << "CUDA error in " << __FILE__ << " at line " << __LINE__ << ": " \
                      << hipGetErrorString(err) << std::endl; \
            exit(EXIT_FAILURE); \
        } \
    } while (0)

int main() {
    float a = 2.0f;
    float b = 1.0f;
    
    int sizes[] = {512, 1024, 2048, 4096, 8192, 16384};

    for (int matSize : sizes) {
        
        std::cout << std::endl;
        float *A, *B;
        int size = matSize * matSize;
        A = (float*)malloc(size * sizeof(float));
        B = (float*)malloc(size * sizeof(float));
        initMat(A, matSize, matSize);

       // hipEvent_t start, stop;
       // CUDA_CHECK(hipEventCreate(&start));
       // CUDA_CHECK(hipEventCreate(&stop));
       // CUDA_CHECK(hipEventRecord(start, 0));
        auto start = std::chrono::high_resolution_clock::now();

        // matTrans(A, a, b, B, matSize, matSize, dim3(32, 32));
        matTransAdv(A, a, b, B, matSize, matSize);

        auto stop = std::chrono::high_resolution_clock::now();
        //  CUDA_CHECK(hipEventRecord(stop, 0));
        //  CUDA_CHECK(hipEventSynchronize(stop));

        //  float elapsedTime;
        //      CUDA_CHECK(hipEventElapsedTime(&elapsedTime, start, stop));
       
        
        //    CUDA_CHECK(hipEventDestroy(start));
        //    CUDA_CHECK(hipEventDestroy(stop));

        free(A);
        free(B);
        
        std::chrono::duration<float, std::milli> duration = stop - start;
        std::cout << "matTrans - Matrix Scalar Add of size " << matSize << "x" << matSize << " took " << duration.count() << " ms." << std::endl;    
    }
    return 0;
}


