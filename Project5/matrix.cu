#include "hip/hip_runtime.h"
#include "matrix.h"
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <hipblas.h>

void initMat(float* matrix, int rows, int cols) {
    for (int i = 0; i < rows * cols; i++) {
        matrix[i] = static_cast<float>(rand()) / static_cast<float>(RAND_MAX);
    }
}

__global__ void matTransKernelAdv(const float* __restrict__ A, float a, float b, float* B, int rows, int cols) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int idy = threadIdx.y + blockIdx.y * blockDim.y;

    int threadIdxInBlock = threadIdx.x + threadIdx.y * blockDim.x;

    extern __shared__ float shared_memory[];
    float* shared_A = shared_memory;

    if (idx < cols && idy < rows) {
        shared_A[threadIdxInBlock] = A[idy * cols + idx];
    }
    
    __syncthreads();



    if (idx < cols && idy < rows) {
        B[idy * cols + idx] = a * shared_A[threadIdxInBlock] + b;
    }
}

void matTrans(const float* A, float a, float b, float* B, int rows, int cols, dim3 threadsPerBlock) {
    float *data_A, *data_B;
    int size = rows * cols * sizeof(float);

    hipMalloc((void**)&data_A, size);
    hipMalloc((void**)&data_B, size);

    hipMemcpy(data_A, A, size, hipMemcpyHostToDevice);

    dim3 blocksPerGrid((cols + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (rows + threadsPerBlock.y - 1) / threadsPerBlock.y);

    matTransKernelAdv<<<blocksPerGrid, threadsPerBlock>>>(data_A, a, b, data_B, rows, cols);
    hipMemcpy(B, data_B, size, hipMemcpyDeviceToHost);

    hipFree(data_A);
    hipFree(data_B);
}

void matTransAdv(const float* A, float a, float b, float* B, int rows, int cols, dim3 threadsPerBlock) {
    float *data_A, *data_B;
    int size = rows * cols * sizeof(float);

    hipMalloc((void**)&data_A, size);
    hipMalloc((void**)&data_B, size);

    hipMemcpy(data_A, A, size, hipMemcpyHostToDevice);

    dim3 blocksPerGrid((cols + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (rows + threadsPerBlock.y - 1) / threadsPerBlock.y);

    int sharedMemSize = threadsPerBlock.x * threadsPerBlock.y * sizeof(float);

    matTransKernelAdv<<<blocksPerGrid, threadsPerBlock, sharedMemSize>>>(data_A, a, b, data_B, rows, cols);
    
    // Check for kernel errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA error in kernel launch: " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }

    hipMemcpy(B, data_B, size, hipMemcpyDeviceToHost);

    hipFree(data_A);
    hipFree(data_B);
}

void cublasSgeamTrans(const float* A, float a, float b, float* B, int rows, int cols) {
    float *d_A, *d_B;
    int size = rows * cols * sizeof(float);

    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);

    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    hipMemset(d_B, 0, size);
    float alpha = a;
    float beta = b;

    hipblasSgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, rows, cols, &alpha, d_A, rows, &beta, d_B, rows, d_B, rows);

    hipMemcpy(B, d_B, size, hipMemcpyDeviceToHost);
    hipFree(d_A);
    hipFree(d_B);
    hipblasDestroy(handle);
}

void cublasSgemmTrans(const float* A, float a, float b, float* B, int rows, int cols) {
    float *d_A, *d_B;
    int size = rows * cols * sizeof(float);

    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);

    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    hipMemset(d_B, 0, size);

    float alpha = a;
    float beta = b;

    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, rows, cols, 1, &alpha, d_A, rows, d_B, rows, &beta, d_B, rows);

    hipMemcpy(B, d_B, size, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipblasDestroy(handle);
}

void matTransCPU(const float* A, float a, float b, float* B, int rows, int cols) {
    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < cols; ++j) {
            int index = i * cols + j;
            B[index] = a * A[index] + b;
        }
    }
}
